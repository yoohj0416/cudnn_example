#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <stdio.h>
#include <iostream>
#include <cmath>

#include "float32.h"

#define IN_DATA_BYTES (IN_SIZE*sizeof(dtype))
#define OUT_DATA_BYTES (OUT_SIZE*sizeof(dtype))

//function to print out error message from cuDNN calls
#define checkCUDNN(exp) \
  { \
    hipdnnStatus_t status = (exp); \
    if(status != HIPDNN_STATUS_SUCCESS) { \
      std::cerr << "Error on line " << __LINE__ << ": " \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE); \
    } \
  } 

int main() {
  hipdnnHandle_t cudnn;
  checkCUDNN(hipdnnCreate(&cudnn));

  hipdnnPoolingDescriptor_t pooling_desc;
  //create descriptor handle
  checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_desc));
  //initialize descriptor
  checkCUDNN(hipdnnSetPooling2dDescriptor(pooling_desc,            //descriptor handle
                                         HIPDNN_POOLING_MAX,       //mode - max pooling
                                         HIPDNN_NOT_PROPAGATE_NAN, //NaN propagation mode
                                         3,                       //window height
                                         3,                       //window width
                                         0,                       //vertical padding
                                         0,                       //horizontal padding
                                         1,                       //vertical stride
                                         1));                     //horizontal stride
  
  hipdnnTensorDescriptor_t in_desc;
  //create input data tensor descriptor
  checkCUDNN(hipdnnCreateTensorDescriptor(&in_desc));
  //initialize input data descriptor 
  checkCUDNN(hipdnnSetTensor4dDescriptor(in_desc,                  //descriptor handle
                                        HIPDNN_TENSOR_NCHW,        //data format
                                        CUDNN_DTYPE,              //data type (precision)
                                        1,                        //number of images
                                        1,                        //number of channels
                                        5,                       //data height 
                                        5));                     //data width

  hipdnnTensorDescriptor_t out_desc;
  //create output data tensor descriptor
  checkCUDNN(hipdnnCreateTensorDescriptor(&out_desc));
  //initialize output data descriptor
  checkCUDNN(hipdnnSetTensor4dDescriptor(out_desc,                 //descriptor handle
                                        HIPDNN_TENSOR_NCHW,        //data format
                                        CUDNN_DTYPE,              //data type (precision)
                                        1,                        //number of images
                                        1,                        //number of channels
                                        3,                        //data height
                                        3));                      //data width

  stype alpha = 1.0f;
  stype beta = 0.0f;
  //GPU data pointers
  dtype *in_data, *out_data;
  //allocate arrays on GPU
  hipMalloc(&in_data,IN_DATA_BYTES);
  hipMalloc(&out_data,OUT_DATA_BYTES);
  //copy input data to GPU array
  hipMemcpy(in_data,input,IN_DATA_BYTES,hipMemcpyHostToDevice);
  //initize output data on GPU
  hipMemset(out_data,0,OUT_DATA_BYTES);

  //Call pooling operator
  checkCUDNN(hipdnnPoolingForward(cudnn,         //cuDNN context handle
                                 pooling_desc,  //pooling descriptor handle
                                 &alpha,        //alpha scaling factor
                                 in_desc,       //input tensor descriptor
                                 in_data,       //input data pointer to GPU memory
                                 &beta,         //beta scaling factor
                                 out_desc,      //output tensor descriptor
                                 out_data));    //output data pointer from GPU memory

  //allocate array on CPU for output tensor data
  dtype *result = (dtype*)malloc(OUT_DATA_BYTES);
  //copy output data from GPU
  hipMemcpy(result,out_data,OUT_DATA_BYTES,hipMemcpyDeviceToHost);

  //loop over and check that the forward pass outputs match expected results (exactly)
  int err = 0;
  for(int i=0; i<OUT_SIZE; i++) {
    if(result[i] != output[i]) {
      std::cout << "Error! Expected " << output[i] << " got " << result[i] << " for idx " << i <<std::endl;
      err++;
    }
  }

  std::cout << "Forward finished with " << err << " errors" << std::endl;

  for(int i=0; i<3; i++)
  {
    for(int j=0; j<3; j++)
    {
      printf("%.2f ", result[i * 3 + j]);
    }
    printf("\n");
  }

  //free CPU arrays
  free(result);

  //free GPU arrays
  hipFree(in_data);
  hipFree(out_data);

  //free cuDNN descriptors
  hipdnnDestroyTensorDescriptor(in_desc);
  hipdnnDestroyTensorDescriptor(out_desc);
  hipdnnDestroyPoolingDescriptor(pooling_desc);
  hipdnnDestroy(cudnn);
  
  return 0;
}